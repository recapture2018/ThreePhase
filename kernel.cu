//verion 1
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include <fstream>

#include <hip/device_functions.h>

using namespace std;
#define SIZE 10240
#define MAXVN 10


__global__ void addKernel(double *c, const double *a, const double *b)
{
    int i = threadIdx.x;
	c[i] = exp(a[i]+b[i]);
}
 
__global__ void simulatedAnnealingKernel(int *route,
										 double * costs,
										 const int solutionLen,
										 const int *demands,
										 const double * distances,
										 const int nodeNum,
										 const int capacities,
										 const int vNum,
										 const int *randInt,
										 const double *randDouble,
										 const double tempParam,
										 const double cr)
{ 
	int depot = threadIdx.x;
	int *bestSolution = route + (depot)*solutionLen;
	__shared__ int solutionArray[SIZE];

	//每个线程分配3*solutonLen*4 Byte 大小的共享内存
	 
	//共享内存变量  
	int *solution = &solutionArray[threadIdx.x*(solutionLen*2+7*MAXVN)]; //solutionLen

    int *curSolution = (int *)&solution[solutionLen];  //solutionLen
	double *dice =(double*)&curSolution[solutionLen];
	double *dis = (double*)&dice[1];
	double *newDis = (double*)&dis[1];
	int *cusIndex = (int*)&newDis[1];
	int *range = (int*)&cusIndex[1];
	int *v = (int*)&range[1];
	int *improvedTryCounter = (int*)&v[1];
    int *demandCounter=(int*)&improvedTryCounter[MAXVN]; //4
	int *routeStart=(int*)&demandCounter[MAXVN];  //6
	double *acc = (double*)&routeStart[MAXVN+1];    //5
	int *r1=v;
	int *r2=range;
	int *inter =cusIndex;

	//寄存器变量
    int maxDemandIndex = 0;
	double demandSum = 0;
    double minCost = 100000;
    int insertPoint = 0;
    int cus=0;
	double cost=0;
	int pre=0;
	int cur=0;
	int i=0;
	int maxDemand=0;
	int nextRand = 0;
	int strategy = 0;
	double temp = tempParam;//cr = 0.001;//tempPara;
	


	//计算距离
    *dis = 0;
	int len = 1;
	for(int i =1; bestSolution[i]!=0; i++){
		  pre = bestSolution[i-1];
		  cur = bestSolution[i];
		  *dis += distances[(pre-1)*nodeNum+cur-1];
		  len ++;
	}

	for(int i =0; i< len; i++){
		solution[i] = bestSolution[i];
	    curSolution[i] = bestSolution[i];
     }
	//检测是否超载
	maxDemandIndex = 0;
	demandSum = 0;
	*v = 0;
	demandCounter[*v] = 0;
	routeStart[*v] = 0;
	for(i = 1;i<len; i++){
	   if (solution[i] == depot+1){
           if( demandCounter[*v] > demandCounter[maxDemandIndex])
                 maxDemandIndex = *v;
			(*v)++;
			demandCounter[*v] = 0;
			routeStart[*v] = i;
			}
		else{
			demandCounter[*v]+= demands[solution[i]-1];
			demandSum += demands[solution[i]-1];
		}
	}
	//判断新解是否满足容量约束
		for(i=0; i<vNum; i++){
		   if(demandCounter[i] > capacities){
		      *dis += capacities;
		   }
		}


	costs[depot] = *dis;
	
	improvedTryCounter[0] =1;
	improvedTryCounter[1]=1;
	
	while(temp > 0.01){
	     //选择策略
	    *dice =randDouble[(nextRand+threadIdx.x*20)%1000];
		nextRand =(nextRand+1)%1000;
		if(*dice <= 0.1+0.8*(improvedTryCounter[0]/(double)(improvedTryCounter[0]+improvedTryCounter[1]))){
			//选择策略1
			strategy = 0;
		    *r1 = 1+randInt[(nextRand+threadIdx.x*20)%1000]%(len-2);
			nextRand = (++nextRand)%1000;
		    *r2 = 1+randInt[(nextRand+threadIdx.x*20)%1000]%(len-2);
			nextRand = (++nextRand)%1000;
			//swap *r1,*r2 in solution
		    *inter = solution[*r1];
			solution[*r1] = solution[*r2];
			solution[*r2] = *inter;
		}
		else{
		    //选择策略2
			//统计各个车俩的负载，根据车辆的负载调整
			strategy = 1;
			
            maxDemandIndex = 0;
			demandSum = 0;
		    *v = 0;
			demandCounter[*v] = 0;
			routeStart[*v] = 0;
			for(i = 1;i<len; i++){
			    if (solution[i] == depot+1){
                    if( demandCounter[*v] > demandCounter[maxDemandIndex])
                         maxDemandIndex = *v;
					(*v)++;
					demandCounter[*v] = 0;
					routeStart[*v] = i;
				}
				else{
					demandCounter[*v]+= demands[solution[i]-1];
				    demandSum += demands[solution[i]-1];
				}
			}
			
			acc[0] = demandCounter[0]/(demandSum+0.1);
			for(i = 1; i< vNum; i++){
			     acc[i] = acc[i-1]+demandCounter[i]/(demandSum+0.1);
			}
			acc[i-1] = 1;

			*dice =randDouble[(nextRand+threadIdx.x*20)%1000];
			nextRand = (++nextRand)%1000;
			*v = 0;
			while(*dice>acc[*v]) (*v)++; //找到对应的车为v
			//从v中抽取一个客户然后将其插入到其他的车的适当路线位置 而客户的位置应该在routeStart[*v]和routeStart[v+1]之间
		    *range = routeStart[*v+1]-routeStart[*v]-1;
			if(*range ==0)
				continue;
		    *cusIndex = routeStart[*v]+1+randInt[(nextRand+threadIdx.x*20)%1000]%*range;
			nextRand = (++nextRand)%1000;


            //按概率选择负载较小的车
		    maxDemand = demandCounter[maxDemandIndex];
			acc[0] = (maxDemand - demandCounter[0])/(vNum*maxDemand - demandSum+0.1);
			for(i =1; i< vNum; i++){
			    acc[i] = acc[i-1]+(maxDemand - demandCounter[i])/(vNum*maxDemand - demandSum+0.1);
			}
			acc[i-1] = 1;

			//更新demandCounter
			demandCounter[*v] -= demands[solution[*cusIndex]-1];

			*dice =randDouble[(nextRand+threadIdx.x*20)%1000];
			nextRand = (++nextRand)%1000;

			*v = 0;
			while(*dice>acc[*v]) (*v)++; //找到对应的车为*v 其范围为routeStart[*v]到routeStart[*v+1]

			//更新demandCounter
			demandCounter[*v] += demands[solution[*cusIndex]-1];

		    minCost = 100000;
		    insertPoint = 0;
		    cus = solution[*cusIndex];
			cost = 0;
			//将*cusIndex对应的客户插入到车辆*v对应的路径中合适的位置
            for(i = routeStart[*v]; i<routeStart[*v+1]; i++){
				   cost = distances[(cus-1)*nodeNum+solution[i]-1]+
					      distances[(cus-1)*nodeNum+solution[i+1]-1]-
						  distances[(solution[i]-1)*nodeNum+solution[i+1]-1];
				   if(cost < minCost){
					   minCost =cost;
					   insertPoint = i;
				   }
			}

		    //将cus从*cusIndex的位置插入到insertPoint的位置
			if( *cusIndex <insertPoint){
			   for(int i = *cusIndex; i <insertPoint; i++){
			         solution[i] = solution[i+1];
			   }
			   solution[insertPoint] = cus;
			}else{
			   for(int i = *cusIndex; i >insertPoint+1; i--){
			          solution[i] = solution[i-1];
			   }
			   solution[insertPoint+1] = cus;
			}
		}

		//计算新解的总距离
	   *newDis = 0;
	    for(i =1; i<len-1;i++){
		 pre = solution[i];
		 cur = solution[i+1];
		*newDis += distances[(pre-1)*nodeNum+cur-1];
	    }
		//判断新解是否满足容量约束
		for(i=0; i<vNum; i++){
		   if(demandCounter[i] > capacities){
		      *newDis += capacities;
		   }
		}

		//如果新解比当前解更优，替换
		if(*newDis < *dis){
			for(i = 0;i< len; i++){
				curSolution[i] = solution[i];
			}
			*dis = *newDis;
			improvedTryCounter[strategy] ++;
			//如果比最优解更优，替换最优解
			if( *newDis < costs[depot]){
				for(i =0; i< len; i++){
				   bestSolution[i] = solution[i];
				}
				costs[depot] = *newDis;
			}
		}else{
			//否则以概率 exp((dis - newDis)/temp)替换
			*dice =randDouble[(nextRand+threadIdx.x*20)%1000];
			nextRand = (++nextRand)%1000;
			if(*dice < exp((*dis - *newDis)/temp)){
			     for(i = 0;i< len; i++){
				   curSolution[i] = solution[i];
			     }
			     *dis = *newDis;
			}else{
				//如果不接受新解，则还原解
			    for(i = 0;i< len; i++){
				   solution[i] = curSolution[i];
			    }
			}
		}
		temp *=1 - cr;
		__syncthreads(); 
	}
}


int main(){
	const int nodeNum = 52;
	const int depotNum = 2;
	const int vehicleNum = 6;
	const int solutionLen = nodeNum - depotNum + vehicleNum + 1;
	int r[depotNum*solutionLen] = {0};
	int demand[nodeNum] = {0};
	double dis[nodeNum*nodeNum]={0};
	double randDouble[1000] = {0};
	int randInt[1000] ={0};
	int capacities = 5000;
	int vNum =6;


	ifstream routeStream("routes.txt");
	ifstream distanceStream("distances.txt");
	ifstream demandStream("demands.txt");
	ifstream rdStream("randDouble.txt");
	ifstream riStream("randInt.txt");
	

	for(int i=0; i<depotNum; i++){
		for(int j =0; j< solutionLen; j++){
			 routeStream>>r[i*solutionLen+j];
		}
	}
	for(int i=0; i<nodeNum; i++){
		demandStream>>demand[i];
		for(int j = 0; j< nodeNum; j++){
		    distanceStream>>dis[i*nodeNum + j];
		}
	}

	for(int i=0; i<1000; i++){
	    rdStream>>randDouble[i];
		riStream>>randInt[i];
	}
	routeStream.close();
	distanceStream.close();
	demandStream.close();
	riStream.close();
	rdStream.close();

	double costs[depotNum]={0};
	hipError_t e;

	int* d_r;
	e = hipMalloc((void**)&d_r,sizeof(int)*depotNum*solutionLen);
    e = hipMemcpy(d_r,r,sizeof(int)*depotNum*solutionLen,hipMemcpyHostToDevice);

	double* d_costs;
	e = hipMalloc((void**)&d_costs,sizeof(double)*depotNum);

	int *d_demand;
	e = hipMalloc((void**)&d_demand,sizeof(int)*nodeNum);
	e = hipMemcpy(d_demand,demand,sizeof(int)*nodeNum,hipMemcpyHostToDevice);

    double *d_dis;
	e = hipMalloc((void**)&d_dis,sizeof(double)*nodeNum*nodeNum);
	e = hipMemcpy(d_dis,dis,sizeof(double)*nodeNum*nodeNum,hipMemcpyHostToDevice);

    int *d_randInt;
    e = hipMalloc((void**)&d_randInt,sizeof(int)*1000);
	e = hipMemcpy(d_randInt,randInt,sizeof(int)*1000,hipMemcpyHostToDevice);

	double *d_randDouble;
	e = hipMalloc((void**)&d_randDouble,sizeof(double)*1000);
	e = hipMemcpy(d_randDouble,randDouble,sizeof(double)*1000,hipMemcpyHostToDevice);
	
	simulatedAnnealingKernel<<<1,depotNum>>>(d_r,d_costs,solutionLen,d_demand,d_dis,nodeNum,capacities,vNum,d_randInt,d_randDouble,100000,0.001);
	
	e = hipMemcpy(r,d_r,sizeof(int)*depotNum*solutionLen,hipMemcpyDeviceToHost);
	e = hipMemcpy(costs,d_costs,sizeof(double)*depotNum,hipMemcpyDeviceToHost);

	hipFree(d_r);
	hipFree(d_costs);
	hipFree(d_demand);
	hipFree(d_dis);
	hipFree(d_randDouble);
	hipFree(d_randInt);
  
 return 0;
}



 